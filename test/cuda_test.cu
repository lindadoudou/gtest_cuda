#include "hip/hip_runtime.h"
//
// Created by xiaolu.lin on 2019/10/25.
//

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include "gtest_cuda.h"

__global__ void test(){
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    printf("%d : __%d__\n", threadId, __LINE__);
}

CUDA_SINGLE_TEST(test);

CUDA_SIMPLE_TEST(test);

CUDA_TEST(test, 32, 2);

__global__ void death_test(){
   assert(0);
}
CUDA_DEATH_TEST(death_test, 1, 1);

__global__ void part_death_test(){
    int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
    assert(threadId > 16);
}
CUDA_DEATH_TEST(part_death_test, 32, 1);