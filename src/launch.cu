//
// Created by xiaolu.lin on 2019/11/4.
//

#include "launch.h"

void launch(void (*func)(), int blocks, int threads, const char * file, const int line)
{
    (*func)<<<blocks, threads>>>();
    hipError_t re = hipDeviceSynchronize();
    checkCudaResult(re, file, line);
}