//
// Created by xiaolu.lin on 2019/11/4.
//
#include <stdio.h>
#include "base.h"

void checkCudaResultMsg(hipError_t cuda_err, const char * file, int line,  const char * message){
    if(hipSuccess != cuda_err)
    {
        printf("%s:%d  %s, ErrorStr[ %s ], ErrorCode[ %d ].\n", file, line, message, hipGetErrorString(cuda_err), cuda_err);
        exit(1);
    }
}

void checkCudaResult(hipError_t cuda_err, const char * file,  int line){
    if(hipSuccess != cuda_err)
    {
        printf("%s: %d ErrorStr[ %s ], ErrorCode[ %d ].\n", file, line, hipGetErrorString(cuda_err), cuda_err);
        exit(1);
    }
}